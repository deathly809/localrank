
#include <util/cudaSync.cuh>

Sync SyncFactory() {
    Sync result;

    CudaTest(hipMalloc(&result.count,sizeof(int)));
    CudaTest(hipMalloc(&result.lock,sizeof(int)));

    CudaTest(hipMemset(result.count,0,sizeof(int)));
    CudaTest(hipMemset(result.lock,result.Unlocked,sizeof(int)));

    return result;
}

void SyncCleanup(Sync & sync) {
    hipFree(sync.count);
    hipFree(sync.lock);

    sync.count  = nullptr;
    sync.lock   = nullptr;
}