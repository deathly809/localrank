
#include <util/cudaUtil.cuh>

/*
 *  Assume that we are working with linear coordinates
 */

/* In this kernel call return my global ID */
__device__ int globalID() {
    return threadIdx.x + blockIdx.x * blockDim.x;
}

__device__ int globalThreadCount() {
    return blockDim.x * gridDim.x;
}

int getMaximumNumberOfBlocks(void) {

    int deviceID;
    CudaTest(hipGetDevice(&deviceID));

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop,deviceID);

    int blocks = 32;
    if(prop.major < 3) {
        blocks = 8;
    }else if(prop.major < 5) {
        blocks = 16;
    }
    return blocks * prop.multiProcessorCount;
}


void CudaBoolean::init() {
    CudaTest(hipMalloc(&value,sizeof(uint)));
}
void CudaBoolean::shutdown() {
    CudaTest(hipFree(value));
}
